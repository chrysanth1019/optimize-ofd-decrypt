#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <openssl/evp.h>
#include <openssl/rand.h>
#include <openssl/err.h>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define MAX_INPUT_LENGTH 1000000 // Max length for each line in encrypted data

__device__ unsigned char hexCharToValue(char hexChar) {
	if (hexChar >= '0' && hexChar <= '9') {
	    return hexChar - '0';
	}
	else if (hexChar >= 'A' && hexChar <= 'F') {
	    return hexChar - 'A' + 10;
	}
	else if (hexChar >= 'a' && hexChar <= 'f') {
	    return hexChar - 'a';
	}
	return 0;
}

__global__ void hex_to_bin_kernel(const char *hex, unsigned char *bin, int hex_len) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < hex_len / 2) {
	unsigned char highNibble = hexCharToValue(hex[idx * 2]);
	unsigned char lowNibble = hexCharToValue(hex[idx * 2 + 1]);
	bin[idx] = (highNibble << 4) | lowNibble;
    }
}

int cuda_hex_to_bin(const char *hex, unsigned char *bin, int bin_size) {
    int hex_len = strlen(hex);
    if (hex_len % 2 != 0 || bin_size < hex_len / 2) {
        return -1;
    }

    char *d_hex;
    unsigned char *d_bin;

    hipMalloc((void **)&d_hex, hex_len * sizeof(char));
    hipMalloc((void **)&d_bin, (hex_len / 2) * sizeof(unsigned char));

    hipMemcpy(d_hex, hex, hex_len * sizeof(char), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (hex_len / 2 + threadsPerBlock - 1) / threadsPerBlock;

    hex_to_bin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_hex, d_bin, hex_len);
    hipDeviceSynchronize();

    hipMemcpy(bin, d_bin, (hex_len / 2) * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_hex);
    hipFree(d_bin);

    return hex_len / 2;
}

// Base58 alphabet
const char *base58_alphabet = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

#define hex2char(c, b) do { \
    if (c >= '0' && c <= '9') { \
        b = c - '0'; \
    } else if (c >= 'A' && c <= 'F') { \
        b = c - 'A' + 10; \
    } else if (c >= 'a' && c <= 'f') { \
        b = c - 'a' + 10; \
    } else { \
	b = 0; \
    } \
} while(0)

// Function to convert hex string to binary data
int hex_to_bin(const char *hex, unsigned char *bin, int bin_size) {
    int hex_len = strlen(hex);
    if (hex_len % 2 != 0) {
        return -1;
    }

    if (bin_size < hex_len / 2) {
        return -1;
    }

    for (int i = 0; i < hex_len / 2; i++) {
#if 1
	unsigned char highNibble, lowNibble;
	hex2char(hex[i * 2], highNibble);
	hex2char(hex[i * 2 + 1], lowNibble);
	bin[i] = (highNibble << 4) | lowNibble;
#else
        sscanf(&hex[i * 2], "%2hhx", &bin[i]);
#endif
    }

    return hex_len / 2;
}

// Check if a block of data is all ASCII
int is_all_ascii(unsigned char *data, int len) {
    for (int i = 0; i < len; i++) {
        if (data[i] < 0 || data[i] > 127) {
            return 0; // Not ASCII
        }
    }
    return 1; // All ASCII
}

// Remove different padding schemes
void remove_padding(unsigned char *data, int *data_len, int padding_type) {
    int padding_len;
    switch (padding_type) {
        case 1: // NoPadding
            break;
    }
}

// Define the AES decryption function with various modes and paddings
int decrypt(unsigned char *ciphertext, int ciphertext_len, unsigned char *password, unsigned char *iv, unsigned char *plaintext, const EVP_CIPHER *cipher, int padding_type, int iterations) {
    EVP_CIPHER_CTX *ctx;
    int len;
    int plaintext_len;
    unsigned char key[32];

    // Derive key using PBKDF2 with user-provided iterations
    if (!PKCS5_PBKDF2_HMAC_SHA1((char *)password, strlen((char *)password), iv, 16, iterations, sizeof(key), key)) {
        return -1;
    }

    // Create and initialize the context
    if (!(ctx = EVP_CIPHER_CTX_new())) {
        return -1;
    }

    // Initialize the decryption operation
    if (1 != EVP_DecryptInit_ex(ctx, cipher, NULL, key, iv)) {
        EVP_CIPHER_CTX_free(ctx);
        return -1;
    }

    // Set padding
    EVP_CIPHER_CTX_set_padding(ctx, padding_type == 6); // Only enable built-in PKCS7 padding for padding_type 6

    // Provide the message to be decrypted and obtain the plaintext output
    if (1 != EVP_DecryptUpdate(ctx, plaintext, &len, ciphertext, ciphertext_len)) {
        EVP_CIPHER_CTX_free(ctx);
        return -1;
    }
    plaintext_len = len;

    // Finalize the decryption
    if (1 != EVP_DecryptFinal_ex(ctx, plaintext + len, &len)) {
        EVP_CIPHER_CTX_free(ctx);
        return -1;
    }
    plaintext_len += len;

    // Clean up
    EVP_CIPHER_CTX_free(ctx);

    // Remove padding if necessary
    if (padding_type != 6) {
        remove_padding(plaintext, &plaintext_len, padding_type);
    }

    return plaintext_len;
}

// Function to sanitize decrypted text (remove weird chars that create new lines or big spaces)
void sanitize_text(unsigned char *text, int len) {
    for (int i = 0; i < len; i++) {
        // If character is not printable, replace it with a space
        if (text[i] < 32 || text[i] > 126) {
            text[i] = ' ';
        }
    }
    text[len] = '\0'; // Null-terminate the string after sanitization
}

// Function to perform the decryption and handle the different modes and paddings
int perform_decryption(unsigned char *ciphertext, int ciphertext_len, unsigned char *password, char *original_hex, int iterations) {
    unsigned char iv[16]; // Initialize IV buffer
    unsigned char decryptedtext[2048000];
    int decryptedtext_len;
    const EVP_CIPHER *modes[] = {
        EVP_aes_256_ofb(), // Use OFB mode
    };
    int paddings[] = {1}; // Only NoPadding

    // Extract IV from the beginning of the ciphertext for modes that require it
    for (int i = 0; i < sizeof(modes) / sizeof(modes[0]); i++) {
        int iv_len = EVP_CIPHER_iv_length(modes[i]);
        unsigned char *ciphertext_copy = ciphertext;
        int ciphertext_len_copy = ciphertext_len;

        if (iv_len > 0) {
            memcpy(iv, ciphertext, iv_len);
            ciphertext_copy += iv_len;
            ciphertext_len_copy -= iv_len;
        } else {
            memset(iv, 0, sizeof(iv)); // Set IV to zero for ECB mode
        }

        for (int j = 0; j < sizeof(paddings) / sizeof(paddings[0]); j++) {
            decryptedtext_len = decrypt(ciphertext_copy, ciphertext_len_copy, (unsigned char *)password, iv, decryptedtext, modes[i], paddings[j], iterations);
            if (decryptedtext_len > 0 && is_all_ascii(decryptedtext, decryptedtext_len)) {
                sanitize_text(decryptedtext, decryptedtext_len);

                // Output hexencoded data | password | decrypted data
                printf("%s|%s|%s\n", original_hex, password, decryptedtext);
                return 0; // Return success for this password
            }
        }
    }

    return -1; // Decryption failed with this password
}

// Main function to handle input from file and passwords
int main(int argc, char *argv[]) {
    if (argc != 4) { // Expect a third argument for iterations
        fprintf(stderr, "Usage: %s <hex input file> <password list file> <iterations>\n", argv[0]);
        return 1;
    }

    char *filename = argv[1];
    char *password_file = argv[2];
    int iterations = atoi(argv[3]); // Convert the third argument to an integer
    if (iterations <= 0) {
        fprintf(stderr, "Invalid iterations value. Must be a positive integer.\n");
        return 1;
    }

    unsigned char encrypted_data[MAX_INPUT_LENGTH / 2]; // To hold up to 100000/2 bytes of decoded binary data
    char hex_line[MAX_INPUT_LENGTH]; // Buffer to hold each hex-encoded line

    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening encrypted data file: %s\n", filename);
        return 1;
    }

    FILE *passwords = fopen(password_file, "r");
    if (!passwords) {
        fprintf(stderr, "Error opening password file: %s\n", password_file);
        fclose(file);
        return 1;
    }

    // Read each line from the encrypted data file
    while (fgets(hex_line, sizeof(hex_line), file)) {
        // Remove newline character if present
        size_t len = strlen(hex_line);
        if (hex_line[len - 1] == '\n') {
            hex_line[len - 1] = '\0';
        }

        // Convert the hex-encoded line to binary
        int encrypted_len;
	clock_t start_time = clock();

#if 1
	encrypted_len = hex_to_bin(hex_line, encrypted_data, sizeof(encrypted_data));
#else
	encrypted_len = cuda_hex_to_bin(hex_line, encrypted_data, sizeof(encrypted_data));
#endif
	printf("time: %f\n", ((double)(clock() - start_time)) / CLOCKS_PER_SEC);

        if (encrypted_len < 0) {
            continue;
        }

        // Reset password file pointer for every new line in the encrypted data file
        fseek(passwords, 0, SEEK_SET);

        // Try each password from the password list
        char password[256];
        while (fgets(password, sizeof(password), passwords)) {
            // Remove newline character if present
            size_t pass_len = strlen(password);
            if (password[pass_len - 1] == '\n') {
                password[pass_len - 1] = '\0';
            }

            // Perform decryption with this password
            if (perform_decryption(encrypted_data, encrypted_len, (unsigned char *)password, hex_line, iterations) == 0) {
                break; // Move to the next encrypted line
            }
        }
    }

    fclose(file);
    fclose(passwords);
    return 0;
}

